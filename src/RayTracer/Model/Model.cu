#include "hip/hip_runtime.h"
#include "Model.cuh"
#define UP_VEC make_float3(0.0f, getRadius(), 0.0f)

CUDA_FUNC float mix_produt(float3 a, float3 b, float3 c)
{
    return dot(cross(a, b), c);
}

CUDA_FUNC Triangle::Triangle(float3 a, float3 b, float3 c, float3 norma, float3 normb, float3 normc)
{
    pos[0] = a;
    pos[1] = b;
    pos[2] = c;
    normal[0] = norma;
    normal[1] = normb;
    normal[2] = normc;
}
CUDA_FUNC Triangle::Triangle(float2 t[3],float3 p[3], float3 norm[3]) {
	vText[0] = t[0];
	vText[1] = t[1];
	vText[2] = t[2];
	pos[0] = p[0];
	pos[1] = p[1];
	pos[2] = p[2];
	normal[0] = norm[0];
	normal[1] = norm[1];
	normal[2] = norm[2];
}
CUDA_FUNC Triangle::Triangle(float3 p[3], float3 norm[3]) {
    pos[0] = p[0];
    pos[1] = p[1];
    pos[2] = p[2];
	normal[0] = norm[0];
    normal[1] = norm[1];
    normal[2] = norm[2];
}
CUDA_FUNC Triangle::Triangle(const float3 p[3], const float3 norm[3]) {
    pos[0] = p[0];
    pos[1] = p[1];
    pos[2] = p[2];
    normal[0] = norm[0];
    normal[1] = norm[1];
    normal[2] = norm[2];
}
CUDA_FUNC Triangle::Triangle(float3 a, float3 b, float3 c, float3 norm[3]) {
    pos[0] = a;
    pos[1] = b;
    pos[2] = c;
    normal[0] = norm[0];
    normal[1] = norm[1];
    normal[2] = norm[2];
}


CUDA_FUNC float3 Triangle::interpolatePosition(float3 sample) const
{
    return transformation(sample.x * pos[0] + sample.y * pos[1] + sample.z * pos[2]);
}


CUDA_FUNC  bool  Triangle::hit(Ray r, IntersectRecord &colideRec) {

    //colideRec.t = -1.0f;
	float3 ta=transformation(pos[0]), tb=transformation(pos[1]), tc= transformation(pos[2]);
	

	float t;
    float3 norma = cross(ta - tc, tb - tc);
    float dot_normal_dir = dot(norma, r.getDir());
    if (fabs(dot_normal_dir) < FLOAT_EPISLON)
        return false;

    t = (-dot(r.getOrigin(), norma) + dot(ta, norma))/ dot_normal_dir;

    float3 pos = r.getPos(t);

    float S = area();
    float s1 = length(cross(pos - ta, pos -tb));
    float s2 = length(cross(pos - tc, pos - ta));
    float s3 = length(cross(pos - tc, pos - tb));
	
	float2 pvText;

    if (fabs(s1 + s2 + s3 - S) > 0.001f)
        return false;

    float m1 = s3 / S, m2 = s2 / S, m3 = 1.0f - m1 - m2;
	pvText = vText[0] * m1 + vText[1] * m2 + vText[2] * m3;
	//Todo: load this to colideRec
    if (t > FLOAT_EPISLON && t < colideRec.t)
    {
        colideRec.material = my_material;
        colideRec.material_type = material_type;
        colideRec.t = t;
        colideRec.normal = normalize(m1 * normal[0] + m2 * normal[1] + m3 * normal[2]);
        colideRec.pos = r.getPos(t);
		colideRec.isLight = false;
        colideRec.tangent =  normalize( cross(colideRec.normal, make_float3(0.3, 0.4, -0.5)));
		return true;
    }

    return false;
}

__host__  bool Mesh::readFile(char * path) {
	ifstream file(path);

	vector<float3> vVertex;
	vector<float2> vText;
	vector<float3> vNorm;
	vector<vector<int3>> vFace;
	if (!file) {
		return false;
	}
	string line;
	while (getline(file, line)) {
		if (line.substr(0, 2) == "vt") {
			istringstream s(line.substr(2));
			float2 v;
			s >> v.x; s >> v.y;

			v.y = -v.y;
			vText.push_back(v);
		}
		else if (line.substr(0, 2) == "vn") {
			istringstream s(line.substr(2));
			float3 v;
			s >> v.x; s >> v.y; s >> v.z;

			vNorm.push_back(v);
		}
		else if (line.substr(0, 1) == "v") {
			istringstream s(line.substr(1));
			float3 v;
			s >> v.x; s >> v.y; s >> v.z;

			vVertex.push_back(v);
		}
		else if (line.substr(0, 1) == "f") {
			vector <int3> face;

			istringstream vtns(line.substr(1));
			string vtn;
			while (vtns >> vtn) {
				int3 vertex;
				replace(vtn.begin(), vtn.end(), '/', ' ');
				istringstream ivtn(vtn);
				if (vtn.find("  ") != string::npos) {
					ivtn >> vertex.x >> vertex.y;

					vertex.x--;
					vertex.y--;
					vertex.z = 0xfff;
				}
				else {
					ivtn >> vertex.x
						>> vertex.y
						>> vertex.z;


					vertex.x--;
					vertex.y--;
					vertex.z--;
				}
				face.push_back(vertex);
			}
			vFace.push_back(face);

		}
		else if (line[0] == '#') {
		}
		else {

		}
	}
	if (vFace.empty())
		return false;
	//vector<Triangle> tempMesh;

	if (vText.size() != 0) {

		hipMalloc((void**)& meshTable, sizeof(Triangle)*(vFace.size() + 1));

		Triangle *temp = (Triangle*)malloc(sizeof(Triangle)*(vFace.size() + 1));
		for (int f = 0; f < vFace.size(); f++) {
			int n = vFace[f].size();

			float3 V[3], N[3];
			float2 T[3];
			for (int v = 0; v < n; v++) {
				int it = vFace[f][v].z;
				if (vText.size() > 0) {
					T[v].x = vText[it].x;
					T[v].y = vText[it].y;
				}
				//	glTexCoord2f(vText[it].x, vText[it].y);

				int in = vFace[f][v].y;
				if (vNorm.size() > 0) {
					V[v].x = vNorm[in].x;
					V[v].y = vNorm[in].y;
					V[v].z = vNorm[in].z;
				}
				int iv = vFace[f][v].x;
				N[v].x = vVertex[iv].x;
				N[v].y = vVertex[iv].y;
				N[v].z = vVertex[iv].z;
				//	glVertex3f(vVertex[iv].x, vVertex[iv].y, vVertex[iv].z);
			}

			Triangle t(T,V,N);

			t.setUpTransformation(transformation);
			temp[f] = t;
			//	glEnd();
		}
		hipMemcpy(meshTable, temp, sizeof(Triangle)*(vFace.size() + 1), hipMemcpyHostToDevice);

	}

	else {

		hipMalloc((void**)& meshTable, sizeof(Triangle)*(vFace.size() + 1));
		Triangle *temp = (Triangle*)malloc(sizeof(Triangle)*(vFace.size() + 1));
		for (int f = 0; f < vFace.size(); f++) {
			int n = vFace[f].size();
			float3 V[3], N[3];
			for (int v = 0; v < n; v++) {
				int in = (vFace[f])[v].y;
				int iv = (vFace[f])[v].x;
				if (vNorm.size() > 0) {
					V[v].x = vNorm[in].x;
					V[v].y = vNorm[in].y;
					V[v].z = vNorm[in].z;
				}
				N[v].x = vVertex[iv].x;
				N[v].y = vVertex[iv].y;
				N[v].z = vVertex[iv].z;
			}
			Triangle t(N,V);
			t.setUpTransformation(transformation);
			temp[f] = t;
		}
		hipMemcpy(meshTable, temp, sizeof(Triangle)*(vFace.size() + 1), hipMemcpyHostToDevice);
		
	}
	number = vFace.size();
	return true;
}


CUDA_FUNC  bool  Mesh::hit(Ray r, IntersectRecord &colideRec) {
    bool ishit = false;
    Triangle t;
//	printf("%d cao\n", number);
    for (int i = 0; i < number; i++) {
        t = *(meshTable + i);
        ishit |= t.hit(r, colideRec);
	}
	
	if (ishit) {
		colideRec.material = my_material;
		colideRec.material_type = material_type;
	}

	return ishit;
}

CUDA_FUNC Quadratic::Quadratic(float3 Coefficient, int Type) {
	coefficient = Coefficient;
	type = Type;
	if (Type == Sphere) {
		if (!(coefficient.x == coefficient.y && coefficient.x == coefficient.z)) {
			return;
		}
	}
}
	
CUDA_FUNC bool Quadratic::setHeight(float Height) {
	if (type == Sphere)
		height = Height;
	else
		return false;
	return true;
}
CUDA_FUNC float3 Quadratic::getCenter() const{
	return float3{ transformation.v[0][3],transformation.v[1][3],transformation.v[2][3] };
}
CUDA_FUNC float Quadratic::getRadius() const{
	if (type == Sphere) {
		return (1/coefficient.x);
	}
    return 0.0f;
}


CUDA_FUNC  bool  Quadratic::hit(Ray r, IntersectRecord &colideRec) {
	if (type == Sphere) {
        float3 center = make_float3(0.0f, 0.0f, 0.0f);
        mat4 inv = inverse(transformation);
        float3 Lorigin = inv(r.getOrigin());

        float4 L4dir = inv(make_float4(r.getDir(), 0));
        float3 Ldir = normalize(make_float3(L4dir.x, L4dir.y, L4dir.z));
        float3 oc = Lorigin - center;
        float dotOCD = dot(Ldir, oc);

        if (dotOCD > 0)
            return false;

        float dotOC = dot(oc, oc);
        float discriminant = dotOCD * dotOCD - dotOC + getRadius()*getRadius();
        float t0, t1;
        if (discriminant < 0)
            return false;
        else if (discriminant < FLOAT_EPISLON)
            t0 = t1 = -dotOCD;
        else {
            discriminant = sqrt(discriminant);
            t0 = -dotOCD - discriminant;
            t1 = -dotOCD + discriminant;
            if (t0 < FLOAT_EPISLON)
                t0 = t1;
        }
        //Need to double-check
        //float3 tangent = normalize(make_float3(transformation(make_float4((cross(cross(Ldir*t0 + Lorigin, make_float3(0.0f, 1.0f / , 0.0f)), Ldir*t0 + Lorigin)),0))));
        //float3 normal = normalize(make_float3(transformation(make_float4(Ldir*t0 + Lorigin, 0))));
        //float3 pos = transformation(Ldir*t0 + Lorigin);

        float3 pos = Ldir * t0 + Lorigin;
        float3 normal = normalize(make_float3(transformation(make_float4(pos, 0.0f))));
        float3 tangent;
        if (pos.x == 0.0f && pos.y == 0.0f)
            tangent = make_float3(transformation(make_float4(0.0f, 0.0f, -1.0f, 0.0f)));
        else tangent = cross(
            normal,
            cross(make_float3(transformation(make_float4(UP_VEC - pos, 0.0f))),
                normal)
            );

        tangent = normalize(tangent);
        pos = transformation(pos);

        //if(t0 > 0.0f)
         //   if (dot(normal, r.getDir()) < 0.0f)
         //      printf("%f\n", dot(normal, r.getDir()));

        if (t0 > FLOAT_EPISLON && t0 < colideRec.t)
        {
            //printf("hhh");
            colideRec.material = my_material;
            colideRec.material_type = material_type;
            colideRec.t = t0;
            colideRec.pos = pos;
            colideRec.normal = normal;
            colideRec.tangent = normalize(tangent);
			colideRec.isLight = false;
            return true;
        }
	} 
	else {
		//TODO-Cylinder.
		                               
	}

	return false;
}


CUDA_FUNC float Triangle::area() const
{
    float3 rpos[3];
    for (int i = 0; i < 3; i++)
        rpos[i] = transformation(pos[i]);

    return length(cross(rpos[2] - rpos[0], rpos[1] - rpos[0]));
}

__host__ bool Model::setUpMaterial(material::MATERIAL_TYPE t, Material *mat)
{
    size_t num;
    switch (t)
    {
    case material::LAMBERTIAN:
        num = sizeof(Lambertian);
        break;
    case material::MATERIAL_NUM:
        //break;
    default:
        num = 0;
        return false;
    }
    material_type = t;
    Material tmp = *mat;
    hipMalloc(&tmp.brdfs, num);
    hipMemcpy(tmp.brdfs, mat->brdfs, num, hipMemcpyHostToDevice);
    auto error =  hipMalloc(&my_material, sizeof(Material));
    error = hipMemcpy(my_material, &tmp, sizeof(Material), hipMemcpyHostToDevice);

    return error == hipSuccess;
}
#include "hip/hip_runtime.h"
#include "Model.cuh"

CUDA_FUNC Triangle::Triangle(float3 p[3], float3 norm[3]) {
    pos[0] = p[0];
    pos[1] = p[1];
    pos[2] = p[2];
    normal[0] = norm[0];
    normal[1] = norm[1];
    normal[2] = norm[2];
}
CUDA_FUNC Triangle::Triangle(const float3 p[3], const float3 norm[3]) {
    pos[0] = p[0];
    pos[1] = p[1];
    pos[2] = p[2];
    normal[0] = norm[0];
    normal[1] = norm[1];
    normal[2] = norm[2];
}
CUDA_FUNC Triangle::Triangle(float3 a, float3 b, float3 c, float3 norm[3]) {
    pos[0] = a;
    pos[1] = b;
    pos[2] = c;
    normal[0] = norm[0];
    normal[1] = norm[1];
    normal[2] = norm[2];
}


CUDA_FUNC float3 Triangle::interpolatePosition(float3 sample) const
{
    return sample.x * pos[0] + sample.y * pos[1] + sample.z * pos[2];
}

CUDA_FUNC Triangle& Triangle::operator=(const Triangle& plus) {
    Triangle t1(plus.pos, plus.normal);
    return(t1);
}
CUDA_FUNC  bool  Triangle::hit(Ray r, IntersectRecord &colideRec) {

    //colideRec.t = -1.0f;

    float3 ab, ac, ap, norm, e, qp;
    float t;
    ab = pos[1] - pos[0];
    ac = pos[2] - pos[0];
    qp = -r.getDir();
    norm = cross(ab, ac);
    float d = dot(qp, norm);
    if (d <= 0.0f) return false;
    ap = r.getOrigin() - pos[0];
    t = dot(ap, norm);
    if (t < 0.0f) return false;
    e = cross(qp, ap);
    float v = dot(ac, e);
    if (v < 0.0f || v > d) return false;
    float w = -dot(ab, e);
    if (w < 0.0f || v + w > d) return false;
    t /= d;

    if (t > FLOAT_EPISLON && t < colideRec.t)
    {
        colideRec.material = my_material;
        colideRec.material_type = material_type;
        colideRec.t = t;
        colideRec.normal = norm;
        colideRec.pos = r.getPos(t);
    }

    return true;
}

__host__  bool Mesh::readFile(char * path) {
	ifstream file(path);

	vector<float3> vVertex;
	vector<float2> vText;
	vector<float3> vNorm;
	vector<vector<int3>> vFace;
	if (!file) {
		return false;
	}
	string line;
	while (getline(file, line)) {
		if (line.substr(0, 2) == "vt") {
			istringstream s(line.substr(2));
			float2 v;
			s >> v.x; s >> v.y;

			v.y = -v.y;
			vText.push_back(v);
		}
		else if (line.substr(0, 2) == "vn") {
			istringstream s(line.substr(2));
			float3 v;
			s >> v.x; s >> v.y; s >> v.z;

			vNorm.push_back(v);
		}
		else if (line.substr(0, 1) == "v") {
			istringstream s(line.substr(1));
			float3 v;
			s >> v.x; s >> v.y; s >> v.z;

			vVertex.push_back(v);
		}
		else if (line.substr(0, 1) == "f") {
			vector <int3> face;

			istringstream vtns(line.substr(1));
			string vtn;
			while (vtns >> vtn) {
				int3 vertex;
				replace(vtn.begin(), vtn.end(), '/', ' ');
				istringstream ivtn(vtn);
				if (vtn.find("  ") != string::npos) {
					ivtn >> vertex.x >> vertex.y;

					vertex.x--;
					vertex.y--;
					vertex.z = 0xfff;
				}
				else {
					ivtn >> vertex.x
						>> vertex.y
						>> vertex.z;


					vertex.x--;
					vertex.y--;
					vertex.z--;
				}
				face.push_back(vertex);
			}
			vFace.push_back(face);

		}
		else if (line[0] == '#') {
		}
		else {

		}
	}
	if (vFace.empty())
		return false;
	vector<Triangle> tempMesh;

	if (vText.size() != 0) {
		for (int f = 0; f < vFace.size(); f++) {
			int n = vFace[f].size();

			float3 V[3], N[3];
			for (int v = 0; v < n; v++) {
				int it = vFace[f][v].z;
				//	glTexCoord2f(vText[it].x, vText[it].y);

				int in = vFace[f][v].y;
				V[v].x = vNorm[in].x;
				V[v].y = vNorm[in].y;
				V[v].z = vNorm[in].z;

				int iv = vFace[f][v].x;
				N[v].x = vVertex[iv].x;
				N[v].y = vVertex[iv].y;
				N[v].z = vVertex[iv].z;
				//	glVertex3f(vVertex[iv].x, vVertex[iv].y, vVertex[iv].z);
			}
			Triangle t(V, N);
			tempMesh.push_back(t);
			//	glEnd();
		}
	}

	else {
		for (int f = 0; f < vFace.size(); f++) {
			int n = vFace[f].size();
			//	glBegin(GL_TRIANGLES);

			float3 V[3], N[3];
			for (int v = 0; v < n; v++) {
				int in = vFace[f][v].y;
				V[v].x = vNorm[in].x;
				V[v].y = vNorm[in].y;
				V[v].z = vNorm[in].z;

				int iv = vFace[f][v].x;
				N[v].x = vVertex[iv].x;
				N[v].y = vVertex[iv].y;
				N[v].z = vVertex[iv].z;
			}
			Triangle t(V, N);
			tempMesh.push_back(t);
			//	glEnd();
		}

	}
	hipMalloc((void**)& meshTable, sizeof(Triangle)*(vFace.size() + 1));
	for (int i = 0; i < vFace.size(); i++) {
		meshTable[i] = tempMesh[i];
	}
	number = vFace.size();
	return true;
}


CUDA_FUNC  bool  Mesh::hit(Ray r, IntersectRecord &colideRec) {
    bool ishit = false;
    for (int i = 0; i < number; i++) {
        ishit |= meshTable[i].hit(r, colideRec);
	}
	return ishit;
}

CUDA_FUNC Quadratic::Quadratic(float3 Coefficient, int Type) {
	coefficient = Coefficient;
	type = Type;
	if (Type == Sphere) {
		if (!(coefficient.x == coefficient.y && coefficient.x == coefficient.z)) {
			return;
		}
	}
}
	
CUDA_FUNC bool Quadratic::setHeight(float Height) {
	if (type == Sphere)
		height = Height;
	else
		return false;
	return true;
}
CUDA_FUNC float3 Quadratic::getCenter() {
	return float3{ transformation.v[0][3],transformation.v[1][3],transformation.v[2][3] };
}
CUDA_FUNC float Quadratic::getRadius(){
	if (type == Sphere) {
		return (1/coefficient.x);
	}
}


CUDA_FUNC  bool  Quadratic::hit(Ray r, IntersectRecord &colideRec) {
	if (type == Sphere) {
		float3 center = getCenter();
		float3 oc = r.getOrigin() - center;
		float dotOCD = dot(r.getDir(), oc);

		if (dotOCD > 0)
			return false;

		float dotOC = dot(oc, oc);
		float discriminant = dotOCD * dotOCD - dotOC + getRadius()*getRadius();
		float t0, t1;
		if (discriminant < 0)
			return false;
		else if (discriminant < FLOAT_EPISLON)
			t0 = t1 = -dotOCD;
		else {
			discriminant = sqrt(discriminant);
			t0 = -dotOCD - discriminant;
			t1 = -dotOCD + discriminant;
			if (t0 < 0)
				t0 = t1;
		}
        if (t0 > FLOAT_EPISLON && t0 < colideRec.t)
        {
            colideRec.material = my_material;
            colideRec.material_type = material_type;
            colideRec.t = t0;
            colideRec.pos = r.getPos(t0);
            colideRec.normal = normalize(colideRec.pos - center);
            return true;
        }
	} 
	else {
		
		                               
	}
}


CUDA_FUNC float Triangle::area() const
{
    float3 rpos[3];
    for (int i = 0; i < 3; i++)
        rpos[i] = transformation(pos[i]);

    return length(cross(rpos[2] - rpos[0], rpos[1] - rpos[0]));
}

__host__ bool Model::setUpMaterial(material::MATERIAL_TYPE t, Material *mat)
{
    size_t num;
    switch (t)
    {
    case material::LAMBERTIAN:
        num = sizeof(Lambertian);
        break;
    case material::MATERIAL_NUM:
        //break;
    default:
        num = 0;
        return false;
    }

    material_type = t;
    auto error =  hipMalloc(&my_material, num);
    error = hipMemcpy(my_material, mat, num, hipMemcpyHostToDevice);

    return error == hipSuccess;
}
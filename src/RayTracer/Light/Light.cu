#include "hip/hip_runtime.h"
#include "Light.cuh"

#ifndef INV_PI
#define INV_PI 0.3183098861837907f
#endif // !INV_PI

CUDA_FUNC PointLight::PointLight(const float3 &position, const float3 &color) : pos(position), illum(color) {}
CUDA_FUNC float3 PointLight::lightIllumi(IntersectRecord &ref, Ray *wi, float2 sample) const 
{
    *wi = Ray(pos, ref.pos - pos);

    //float t1 = length(ref.pos - pos);
    ref.pdf_light = 1.0f;
    return illum / fmaxf(0.001f, dot(ref.pos - pos, ref.pos - pos));
}

CUDA_FUNC float3 PointLight::getPower(float3 bound_length) const
{
    return 4.0f * M_PI * illum;
}

DirectionalLight::DirectionalLight(const float3 &direction, const float3 &color) : dir(normalize(direction)), illum(color) {}

float3 DirectionalLight::lightIllumi(IntersectRecord &ref, Ray *wi, float2 sample) const
{
    *wi = Ray(ref.pos - dir * 10000.0f, dir);
    ref.pdf_light = 1.0f;
    return illum;
}

CUDA_FUNC float3 DirectionalLight::getPower(float3 bound_length) const
{
    return illum * dot(bound_length, bound_length) * 0.25f * M_PI;
}

CUDA_FUNC TriangleLight::TriangleLight(float3 posa, float3 posb, float3 posc, const float3& light_color, bool two) : illum(light_color), two_side(two){
    pos[0] = posa;
    pos[1] = posb;
    pos[2] = posc;
    normal = normalize(cross(pos[2] - pos[0], pos[1] - pos[0]));
}

CUDA_FUNC float3 TriangleLight::interpolatePosition(float3 tri_sample) const
{
    return tri_sample.x * pos[0] + tri_sample.y * pos[1] + tri_sample.z * pos[2];
}
//Incompleted
CUDA_FUNC float3 TriangleLight::lightIllumi(IntersectRecord &ref, Ray *wi, float2 sample) const
{
    float sq_x = sqrtf(sample.x);
    float3 tri_sample = make_float3(1 - sq_x, sample.y * sq_x, 0);
    tri_sample.z = 1.0f - tri_sample.x - tri_sample.y;
    float3 pos = interpolatePosition(tri_sample);

    if (wi == nullptr)
        return BLACK;
    else *wi = Ray(pos, ref.pos - pos);

    float t = ref.t;
    ref.t = length(ref.pos - pos);
    //bugs here
    ref.pdf_light = PDF(ref, wi ->getDir());
    ref.t = t;
    return illum;
}


CUDA_FUNC bool TriangleLight::hit(Ray &r, IntersectRecord &rec)
{

    float t;
    float3 normal = cross(pos[0] - pos[2], pos[1] - pos[2]);
    float dot_normal_dir = dot(normal, r.getDir());
    if (fabs(dot_normal_dir) < FLOAT_EPISLON)
        return false;

    t = (-dot(r.getOrigin(), normal) + dot(pos[0], normal)) / dot_normal_dir;

    float3 rpos = r.getPos(t);

    float S = area();
    float s1 = length(cross(rpos - pos[0], rpos - pos[1]));
    float s2 = length(cross(rpos - pos[2], rpos - pos[0]));
    float s3 = length(cross(rpos - pos[2], rpos - pos[1]));


    if (fabs(s1 + s2 + s3 - S) > 0.001f)
        return false;

    float m1 = s3 / S, m2 = s2 / S, m3 = 1.0f - m1 - m2;

    if (t > FLOAT_EPISLON && t < rec.t)
    {
        rec.material = my_material;
        rec.material_type = material_type;
        rec.t = t;
        rec.normal = two_side ? (dot(normal, r.getDir()) > 0 ? -normal : normal) : normal;
        rec.pos = r.getPos(t);
        rec.isLight = true;
        rec.tangent = normalize((pos[1] - pos[0]) * m2 + (pos[2] - pos[0]) * m3);
        return true;
    }

    return false;
}

CUDA_FUNC float3 TriangleLight::getPower(float3 bound_lenght) const
{
    return illum * area() * M_PI * (two_side ? 2.0f : 1.0f);
}

CUDA_FUNC bool TriangleLight::setUpMaterial(material::MATERIAL_TYPE t, Material *mat)
{
    size_t num;
    switch (t)
    {
    case material::LAMBERTIAN:
        num = sizeof(Lambertian);
        break;
    case material::MATERIAL_NUM:
        //break;
    default:
        num = 0;
        return false;
    }

    material_type = t;
    Material tmp = *mat;
    hipMalloc(&tmp.brdfs, num);
    hipMemcpy(tmp.brdfs, mat->brdfs, num, hipMemcpyHostToDevice);
    auto error = hipMalloc(&my_material, sizeof(Material));
    error = hipMemcpy(my_material, &tmp, sizeof(Material), hipMemcpyHostToDevice);

    return error == hipSuccess;
}
//sample = (theta, phi)
CUDA_FUNC float3 EnvironmentLight::lightIllumi(IntersectRecord &ref, Ray *wi, float2 sample)
{
    float theta = sample.x * M_PI, phi = 2.0f * sample.y * M_PI;
    *wi = Ray(wi->getOrigin(), make_float3(sin(theta) * sin(phi),cos(theta), sin(theta) * cos(phi)));
    int idx = sample.x * height * width + sample.y * width;
    idx *= 3;
    //May use MIPMAP
    return make_float3(img[idx], img[idx + 1], img[idx + 2]);
}

CUDA_FUNC float EnvironmentLight::PDF(IntersectRecord rec, const float3 &wi)const
{
    float3 k = normalize(wi);
    float phi = 0.0f;
    float theta = acos(k.y);
    float sint = sin(theta);
    float phi = atan2f(k.x, k.z);

    if(sint < 0.0001f)
        return 0.0f;

    return distribution->PDF(make_float2(phi, theta)) / (2.0f * M_PI * M_PI * sint);
}

CUDA_FUNC float3 EnvironmentLight::getLe(Ray &r) const
{
    return L(r.getDir(), nullptr);
}

CUDA_FUNC float3 EnvironmentLight::L(const float3 &r, IntersectRecord *rec = nullptr) const
{
    float3 k = normalize(r);
    float phi = 0.0f;
    int idx;
    float theta = acos(k.y);
    float sint = sin(theta);
    if (sint > 0.001f)
    {
        float phi = atan2f(r.x, r.z);
        if (phi < 0.0f)
            phi += 2.0f * M_PI;
        idx = theta  * INV_PI * height * width + phi * INV_PI  * width * 0.5f;
        idx *= 3;
    }
    else
        idx = 0;

    return make_float3(img[idx], img[idx + 1], img[idx + 2]);
}
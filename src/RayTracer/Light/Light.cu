#include "hip/hip_runtime.h"
#include "Light.cuh"

CUDA_FUNC PointLight::PointLight(const float3 &position, const float3 &color) : pos(position), illum(color) {}
CUDA_FUNC float3 PointLight::lightIllumi(IntersectRecord &ref, Ray *wi, float2 sample) const 
{
    *wi = Ray(pos, ref.pos - pos);

    float t1 = length(ref.pos - pos);

    return illum / fmaxf(0.001f, dot(ref.pos - pos, ref.pos - pos));
}

DirectionalLight::DirectionalLight(const float3 &direction, const float3 &color) : dir(normalize(direction)), illum(color) {}

float3 DirectionalLight::lightIllumi(IntersectRecord &ref, Ray *wi, float2 sample) const
{
    *wi = Ray(ref.pos - dir * 10000.0f, dir);

    return illum;
}

CUDA_FUNC float3 PointLight::getDir(float3 pos0 = make_float3(0.0f, 0.0f, 0.0f), float2 sample = make_float2(0.0f, 0.0f)) const
{
    return pos - pos0;
}

CUDA_FUNC float3 DirectionalLight::getDir(float3 pos0 = make_float3(0.0f, 0.0f, 0.0f), float2 sample = make_float2(0.0f, 0.0f)) const
{
    return dir;
}

CUDA_FUNC float3 TriangleLight::getDir(float3 pos0 = make_float3(0.0f, 0.0f, 0.0f), float2 sample = make_float2(0.0f, 0.0f)) const
{
    static float3 pos;
    pos = tri.interpolatePosition(make_float3(sample, 1.0f - sample.x - sample.y));
    return pos - pos0;
}

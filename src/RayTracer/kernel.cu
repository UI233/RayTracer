#include "hip/hip_runtime.h"
﻿#define WIDTH 800
#define HEIGHT 600
#define WIDTH_PER_BLOCK  80
#define HEIGHT_PER_BLOCK  60
#define NUM 16
#define MAX_DEPTH 10
#define SAMPLE 4
#define WARP_SIZE 32
#include <glut/gl3w.h>
#include <Windows.h>
#include <stdio.h>
#include "surface_functions.h"
#include <vector_functions.hpp>
#include <cuda_gl_interop.h>
#include <hip/device_functions.h>
#include "core/PathTracer.cuh"
#include "Camera/Camera.cuh"
#include <glut/glfw3.h>
#include "Shader/myShader.h"
#include "core/PathTracer.cuh"
#include "Ray/Ray.cuh"

__constant__ Camera globalCam;
hiprandState *state;
float *data_tmp;
Scene *sce;
int *cnt;

int thread_num = 10 * 10 * 32;
__global__ void test(hipSurfaceObject_t surface, Scene *scene, hiprandState *state, float* data_tmp, int *cnt);
void display();
__device__ void computeTexture();
bool renderScene(bool);
GLuint initGL();
GLFWwindow* glEnvironmentSetup();
bool initCUDA(GLuint glTex);
void test_for_initialize_scene();

GLuint tex;
GLuint prog;
hipGraphicsResource *cudaTex;
hipSurfaceObject_t texture_surface;

int main(int argc, char **argv)
{
    GLFWwindow *window = glEnvironmentSetup();
    bool changed = true, sta = true;
    tex = initGL();
    initCUDA(tex);
    test_for_initialize_scene();
    //loadModels();

    while (!glfwWindowShouldClose(window) && sta)
    {
        sta = renderScene(changed);
        //changed = false;
        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    glfwDestroyWindow(window);
    glfwTerminate();
    hipFree(state);
    hipFree(data_tmp);
    return 0;
}

bool renderScene(bool changed)
{
    dim3 block_size;

    block_size.x = WIDTH / WIDTH_PER_BLOCK;
    block_size.y = HEIGHT / HEIGHT_PER_BLOCK;
    block_size.z = 1;
    //ToDo: the state should be an array
    test <<<block_size, 32>>> (texture_surface, sce, state, data_tmp, cnt);
    auto error = hipDeviceSynchronize();
    //hipDeviceSynchronize();
    display();

    //test << < WIDTH, HEIGHT >> > (texture_surface, 1.0f);
    //hipDeviceSynchronize();
    //display();

    return error == hipSuccess;
}

GLuint initGL()
{
    //The position of the quad which covers the full screen
    static float vertices[6][2] = {
        -1.0f, 1.0f,
        -1.0f, -1.0f,
        1.0f, 1.0f,
        1.0f, 1.0f,
        -1.0f, -1.0f,
        1.0f, -1.0f
    };
    GLuint tex;
    //initialize the empty texture
    //and set the parameter for it
    glGenTextures(1, &tex);
    glBindTexture(GL_TEXTURE_2D, tex);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, WIDTH, HEIGHT, 0, GL_RGBA, GL_FLOAT, nullptr);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

    GLuint buffer;
    GLuint vao;
    //Push the vertices information into the vertex arrayy
    glCreateBuffers(1, &buffer);
    glCreateVertexArrays(1, &vao);

    glBindBuffer(GL_ARRAY_BUFFER, buffer);
    glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);
    glVertexAttribPointer(1, 2, GL_FLOAT, NULL, NULL, nullptr);

    glEnableVertexAttribArray(1);

    //Initialize the OpenGL shaders and program
    prog = glCreateProgram();
    Shader vertex, frag;

    vertex.LoadFile("./Shader/texture.vert");
    frag.LoadFile("./Shader/texture.frag");
    vertex.Load(GL_VERTEX_SHADER, prog);
    frag.Load(GL_FRAGMENT_SHADER, prog);
    glLinkProgram(prog);
    glBindTexture(GL_TEXTURE_2D, 0);

    return tex;
}

GLFWwindow* glEnvironmentSetup()
{
    glfwInit();
    

    GLFWwindow *window = glfwCreateWindow(WIDTH, HEIGHT, "test", NULL, NULL);
    glfwMakeContextCurrent(window);

    gl3wInit();

    return window;
}

bool initCUDA(GLuint glTex)
{
    auto error = cudaGLSetGLDevice(0);
    error = hipGraphicsGLRegisterImage(&cudaTex, tex, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore);
    error = hipGraphicsMapResources(1, &cudaTex, 0);

    hipArray_t texArray;
    error = hipGraphicsSubResourceGetMappedArray(&texArray, cudaTex, 0, 0);

    hipResourceDesc dsc;
    dsc.resType = hipResourceTypeArray;
    dsc.res.array.array = texArray;

    error = hipCreateSurfaceObject(&texture_surface, &dsc);
    
    Camera cam(make_float3(0.0f, 0.0f, 0.0f), make_float3(0.0f, 0.0f, -1.0f), 2.0f, 1.00f, 100.0f,
        make_int2(WIDTH / 2, HEIGHT / 2), make_float3(0.0f, 1.0f, 0.0f));

    hipMalloc(&state, sizeof(hiprandState) * thread_num);
    error = hipMemcpyToSymbol(HIP_SYMBOL(globalCam), &cam, sizeof(Camera));

    error = hipMalloc(&data_tmp, sizeof(float) * HEIGHT * WIDTH * 3);
    error = hipMemset(data_tmp, 0, sizeof(float) * HEIGHT * WIDTH * 3);

    error = hipMalloc(&cnt, sizeof(int));
    error = hipMemset(cnt, 0, sizeof(int));
    return error == hipSuccess;
}

void display()
{
    glUseProgram(prog);
    glBindTexture(GL_TEXTURE_2D, tex);
    glClear(GL_COLOR_BUFFER_BIT);
    glDrawArrays(GL_TRIANGLES, 0, 6);
}


__global__ void test(hipSurfaceObject_t surface, Scene *scene, hiprandState *state, float* data_tmp, int *cnt)
{
    __shared__ StratifiedSampler<TWO_FOR_SHARED> sampler;

    int idx = (blockIdx.x * blockDim.y + blockIdx.y) * 32 + threadIdx.x;


    //To do: use time to update the seed
    hiprand_init(1774, idx,0, &state[idx]);

    if (threadIdx.x == 0)
    {
       sampler = StratifiedSampler<TWO_FOR_SHARED>(state + idx);
    }

    __syncthreads();
    hiprandState *rstate = state + idx;

    Ray r;
    float3 tmp;
    int stx = blockIdx.x * WIDTH_PER_BLOCK, sty = blockIdx.y * HEIGHT_PER_BLOCK;

    StratifiedSampler<TWO> sampler_light(16, rstate);
    StratifiedSampler<TWO> sampler_surface(16, rstate);
    StratifiedSampler<ONE> p(8, rstate);

    float2 offset = make_float2(0.0f, 0.0f) ;
    int x, y;
    for(int i = 0; i < WIDTH_PER_BLOCK; i++)
        for (int j = 0; j < HEIGHT_PER_BLOCK; j++)
        {
            offset = sampler(threadIdx.x);
            r = globalCam.generateRay(i + stx + offset.x - WIDTH / 2, j + sty + offset.y - HEIGHT / 2);
            tmp = pathTracer(r, *scene, sampler_surface, sampler_light, p, rstate);

            idx = (j + sty) * WIDTH + i + stx;
            idx = idx * 3;
            data_tmp[idx] += tmp.x;
            data_tmp[idx + 1] += tmp.y;
            data_tmp[idx + 2] += tmp.z;

            //sampler_surface.regenerate(rstate);
           // sampler_light.regenerate(rstate);
           // p.regenerate(rstate);
        }
    __syncthreads();

    idx = 3 * (sty * WIDTH + stx);
    //Write to Texture
    if(threadIdx.x == 0)
        for (int i = 0; i < WIDTH_PER_BLOCK; i++)
            for (int j = 0; j < HEIGHT_PER_BLOCK; j++)
            {
                x = i + stx;
                y = j + sty;
                idx = y * WIDTH + x;
                idx = idx * 3;
                //Bug here: should take average value
                surf2Dwrite(make_float4(data_tmp[idx] / 32.0f, 
                    data_tmp[idx + 1] / 32.0f, data_tmp[idx + 2] / 32.0f, 1.0f), surface, sizeof(float4) * x , y);
            }


}

void test_for_initialize_scene()
{
    Scene scene;
    int lz[light::TYPE_NUM] = {1,0,0}, ms[model::TYPE_NUM] = {0,0,1};
    int mat_type[] = { material::LAMBERTIAN };
    Lambertian lamb(make_float3(0.7f, 0.8f, 0.4f));
    Material m(&lamb, material::LAMBERTIAN);
    Quadratic q(make_float3(0.3f, 0.0f, 0.0f), Sphere);
    q.setUpTransformation(
        mat4(1.0f, 0.0f, 0.0f, 0.0f,
             0.0f, 1.0f, 0.0f, 0.0f,
            0.0f, 0.0f,1.0f, -10.0f,
            0.0f,0.0f,0.0f,1.0f)
    );
    PointLight pl(make_float3(-8.0f, 0.0f, 0.0f), make_float3(300.0, 350.0f, 300.0f));

    scene.initializeScene(
        lz, ms, &pl, nullptr, nullptr, nullptr, nullptr,
        &q, mat_type, &m
    );

    hipMalloc(&sce, sizeof(Scene));
    hipMemcpy(sce, &scene, sizeof(Scene), hipMemcpyHostToDevice);
//    cas<<<1,1>>>(sce);
}
#include "hip/hip_runtime.h"
﻿#define HEIGHT 200
#define WIDTH 200
#define NUM 16
#define MAX_DEPTH 10
#define SAMPLE 4
#define WARP_SIZE 32

#include <glut/gl3w.h>
#include <Windows.h>
#include <stdio.h>
#include "surface_functions.h"
#include <vector_functions.hpp>
#include <cuda_gl_interop.h>
#include <hip/device_functions.h>
#include "Model/Model.cuh"
#include "Camera/Camera.cuh"
#include <glut/glfw3.h>
#include "Shader/myShader.h"

__global__ void test(hipSurfaceObject_t surface);
void display();
__device__ void computeTexture();
bool renderScene(bool);
GLuint initGL();
GLFWwindow* glEnvironmentSetup();
bool initCUDA(GLuint glTex);

GLuint tex;
GLuint prog;
hipGraphicsResource *cudaTex;
hipSurfaceObject_t texture_surface;

int main(int argc, char **argv)
{
    GLFWwindow *window = glEnvironmentSetup();
    bool changed = true, state = true;
    tex = initGL();
    initCUDA(tex);
    //loadModels();

    while (!glfwWindowShouldClose(window) && state)
    {
        state = renderScene(changed);
        //changed = false;
        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    glfwDestroyWindow(window);
    glfwTerminate();
    return 0;
}

bool renderScene(bool changed)
{
    test <<< WIDTH, HEIGHT >>> (texture_surface);
    auto error = hipDeviceSynchronize();

    display();

    //test << < WIDTH, HEIGHT >> > (texture_surface, 1.0f);
    //hipDeviceSynchronize();
    //display();

    return error == hipSuccess;
}

GLuint initGL()
{
    //The position of the quad which covers the full screen
    static float vertices[6][2] = {
        -1.0f, 1.0f,
        -1.0f, -1.0f,
        1.0f, 1.0f,
        1.0f, 1.0f,
        -1.0f, -1.0f,
        1.0f, -1.0f
    };
    GLuint tex;
    //initialize the empty texture
    //and set the parameter for it
    glGenTextures(1, &tex);
    glBindTexture(GL_TEXTURE_2D, tex);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, WIDTH, HEIGHT, 0, GL_RGBA, GL_FLOAT, nullptr);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

    GLuint buffer;
    GLuint vao;
    //Push the vertices information into the vertex arrayy
    glCreateBuffers(1, &buffer);
    glCreateVertexArrays(1, &vao);

    glBindBuffer(GL_ARRAY_BUFFER, buffer);
    glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);
    glVertexAttribPointer(1, 2, GL_FLOAT, NULL, NULL, nullptr);

    glEnableVertexAttribArray(1);

    //Initialize the OpenGL shaders and program
    prog = glCreateProgram();
    Shader vertex, frag;

    vertex.LoadFile("./Shader/texture.vert");
    frag.LoadFile("./Shader/texture.frag");
    vertex.Load(GL_VERTEX_SHADER, prog);
    frag.Load(GL_FRAGMENT_SHADER, prog);
    glLinkProgram(prog);
    glBindTexture(GL_TEXTURE_2D, 0);

    return tex;
}

GLFWwindow* glEnvironmentSetup()
{
    glfwInit();
    

    GLFWwindow *window = glfwCreateWindow(WIDTH, HEIGHT, "test", NULL, NULL);
    glfwMakeContextCurrent(window);

    gl3wInit();

    return window;
}

bool initCUDA(GLuint glTex)
{
    auto error = cudaGLSetGLDevice(0);
    error = hipGraphicsGLRegisterImage(&cudaTex, tex, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore);
    error = hipGraphicsMapResources(1, &cudaTex, 0);

    hipArray_t texArray;
    error = hipGraphicsSubResourceGetMappedArray(&texArray, cudaTex, 0, 0);

    hipResourceDesc dsc;
    dsc.resType = hipResourceTypeArray;
    dsc.res.array.array = texArray;

    error = hipCreateSurfaceObject(&texture_surface, &dsc);

    return error == hipSuccess;
}

void display()
{
    glUseProgram(prog);
    glBindTexture(GL_TEXTURE_2D, tex);
    glClear(GL_COLOR_BUFFER_BIT);
    glDrawArrays(GL_TRIANGLES, 0, 6);
}

__global__ void test(hipSurfaceObject_t surface)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    
    int y = idx / WIDTH;
    int x = idx % WIDTH;

    float4 data = make_float4((float) x / WIDTH, (float) y / HEIGHT, 0.0f, 1.0f);


    surf2Dwrite(data, surface, x * sizeof(float4), y);
}

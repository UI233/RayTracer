#include "hip/hip_runtime.h"
﻿#define WIDTH 280
#define HEIGHT 210
#define WIDTH_PER_BLOCK  40
#define HEIGHT_PER_BLOCK  30
#define NUM 16
#define MAX_DEPTH 10
#define SAMPLE 4
#define WARP_SIZE 32
#include <glut/gl3w.h>
#include <Windows.h>
#include <stdio.h>
#include "surface_functions.h"
#include <vector_functions.hpp>
#include <cuda_gl_interop.h>
#include <hip/device_functions.h>
#include "core/PathTracer.cuh"
#include "Camera/Camera.cuh"
#include <glut/glfw3.h>
#include "Shader/myShader.h"
#include "core/PathTracer.cuh"
#include "Ray/Ray.cuh"
#include <iostream>
#include <ctime>

__constant__ Camera globalCam;
hiprandState *state;
float *data_tmp;
Scene *sce;
int *cnt;
float img[3 * WIDTH * HEIGHT];

int thread_num = WIDTH * HEIGHT;
__global__ void test(hipSurfaceObject_t surface, Scene *scene, hiprandState *state, float* data_tmp, int *cnt);

__global__ void initial(hiprandState *state, int *time)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(idx , idx, 0, state + idx);
}

__global__ void debug(hipSurfaceObject_t surface, Scene *scene,float *data_tmp, hiprandState *state, int *cnt)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
        ++*cnt;

    __syncthreads();
    int x = threadIdx.x, y = blockIdx.x;
    int idx = y * blockDim.x + x;
    hiprandState *rstate = state + idx;
    //hiprand_init(1234, idx, 0, rstate);//Produce many noises after initialization
    IntersectRecord rec;

    StratifiedSampler<TWO> sampler_light(16, rstate);
    StratifiedSampler<TWO> sampler_surface(16, rstate);
    StratifiedSampler<ONE> p(8, rstate);
    Ray r = globalCam.generateRay(x - WIDTH / 2, y - HEIGHT / 2);
    float3 tmp = make_float3(1.0f, 1.0f, 1.0f);
    
    //if(x == WIDTH / 2 && y == HEIGHT / 2)
    tmp = pathTracer(r, *scene,sampler_light,sampler_surface,p ,state + idx);
    float frac1 = (float)(*cnt - 1) / *cnt, frac2 = 1.0f / *cnt;
    idx *= 3;
    data_tmp[idx] = frac1 * data_tmp[idx] + frac2 * tmp.x;
    data_tmp[idx + 1] = frac1 * data_tmp[idx + 1] + frac2 * tmp.y;
    data_tmp[idx + 2] = frac1 * data_tmp[idx + 2] + frac2 * tmp.z;

    surf2Dwrite(make_float4(data_tmp[idx], data_tmp[idx + 1], data_tmp[idx + 2], 1.0f), surface, x * sizeof(float4), y);
}

void display();
__device__ void computeTexture();
bool renderScene(bool);
GLuint initGL();
GLFWwindow* glEnvironmentSetup();
bool initCUDA(GLuint glTex);
void test_for_initialize_scene();

GLuint tex;
GLuint prog;
hipGraphicsResource *cudaTex;
hipSurfaceObject_t texture_surface;

int main(int argc, char **argv)
{
    GLFWwindow *window = glEnvironmentSetup();
    bool changed = true, sta = true;
    tex = initGL();
    initCUDA(tex);
    test_for_initialize_scene();
    //loadModels();
    int t = time(NULL);
    int *p;
    hipMalloc(&p, sizeof(int));
    hipMemcpy(p, &t, sizeof(int), hipMemcpyHostToDevice);
    initial << <HEIGHT, WIDTH >> > (state, p);
    auto error = hipDeviceSynchronize();
    while (!glfwWindowShouldClose(window) && sta && changed)
    {
        sta = renderScene(changed);
        //changed = false;
        glfwSwapBuffers(window);
        glfwPollEvents();
    }
    //for (int i = 0; i < 1; i++)
    //{
    //    renderScene(true);
    ////}
    //hipMemcpy(img, data_tmp, 3 * WIDTH * HEIGHT * sizeof(float), hipMemcpyDeviceToHost);
    //FILE * cas = fopen("test.ppm", "w");
    //fprintf(cas, "P3\n%d %d\n255\n", WIDTH, HEIGHT);

    //for (int j = HEIGHT - 1; j >= 0; j--)
    //    for (int i = 0; i < WIDTH; i++)
    //    {
    //        int idx = (WIDTH * j + i) * 3;
    //        int r = img[idx] * 255, g = img[idx + 1] * 255, b = img[idx + 2] * 255;
    //        r = min(255, r);
    //        g = min(255, g);
    //        b = min(255, b);

    //        fprintf(cas, "%d %d %d\n", r, g, b);
    //        //idx += 3;
    //    }
    //fclose(cas);
    glfwDestroyWindow(window);
    glfwTerminate();
    hipFree(state);
    hipFree(data_tmp);
    return 0;
}

bool renderScene(bool changed)
{
    dim3 block_size;

    block_size.x = WIDTH / WIDTH_PER_BLOCK;
    block_size.y = HEIGHT / HEIGHT_PER_BLOCK;
    block_size.z = 1;
    //ToDo: the state should be an array
    hipError_t error;
    //if (changed)
    //{
        debug << <HEIGHT, WIDTH >> > (texture_surface, sce, data_tmp,state, cnt);
        error = hipDeviceSynchronize();
    //}
    display();

    int idx = 0;
    

    return true;
}

GLuint initGL()
{
    //The position of the quad which covers the full screen
    static float vertices[6][2] = {
        -1.0f, 1.0f,
        -1.0f, -1.0f,
        1.0f, 1.0f,
        1.0f, 1.0f,
        -1.0f, -1.0f,
        1.0f, -1.0f
    };
    GLuint tex;
    //initialize the empty texture
    //and set the parameter for it
    glGenTextures(1, &tex);
    glBindTexture(GL_TEXTURE_2D, tex);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, WIDTH, HEIGHT, 0, GL_RGBA, GL_FLOAT, nullptr);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

    GLuint buffer;
    GLuint vao;
    //Push the vertices information into the vertex arrayy
    glCreateBuffers(1, &buffer);
    glCreateVertexArrays(1, &vao);

    glBindBuffer(GL_ARRAY_BUFFER, buffer);
    glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);
    glVertexAttribPointer(1, 2, GL_FLOAT, NULL, NULL, nullptr);

    glEnableVertexAttribArray(1);

    //Initialize the OpenGL shaders and program
    prog = glCreateProgram();
    Shader vertex, frag;

    vertex.LoadFile("./Shader/texture.vert");
    frag.LoadFile("./Shader/texture.frag");
    vertex.Load(GL_VERTEX_SHADER, prog);
    frag.Load(GL_FRAGMENT_SHADER, prog);
    glLinkProgram(prog);
    glBindTexture(GL_TEXTURE_2D, 0);

    return tex;
}

GLFWwindow* glEnvironmentSetup()
{
    glfwInit();
    

    GLFWwindow *window = glfwCreateWindow(WIDTH, HEIGHT, "test", NULL, NULL);
    glfwMakeContextCurrent(window);

    gl3wInit();

    return window;
}

bool initCUDA(GLuint glTex)
{
    auto error = cudaGLSetGLDevice(0);
    error = hipGraphicsGLRegisterImage(&cudaTex, tex, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore);
    error = hipGraphicsMapResources(1, &cudaTex, 0);

    hipArray_t texArray;
    error = hipGraphicsSubResourceGetMappedArray(&texArray, cudaTex, 0, 0);

    hipResourceDesc dsc;
    dsc.resType = hipResourceTypeArray;
    dsc.res.array.array = texArray;

    error = hipCreateSurfaceObject(&texture_surface, &dsc);
    
    Camera cam(make_float3(0.0f, 0.0f, 0.0f), make_float3(0.0f, 0.0f, -1.0f), 2.0f, 1.00f, 100.0f,
        make_int2(WIDTH / 2, HEIGHT / 2), make_float3(0.0f, 1.0f, 0.0f));

    hipMalloc(&state, sizeof(hiprandState) * thread_num);
    error = hipMemcpyToSymbol(HIP_SYMBOL(globalCam), &cam, sizeof(Camera));

    error = hipMalloc(&data_tmp, sizeof(float) * HEIGHT * WIDTH * 3);
    error = hipMemset(data_tmp, 0, sizeof(float) * HEIGHT * WIDTH * 3);

    error = hipMalloc(&cnt, sizeof(int));
    error = hipMemset(cnt, 0, sizeof(int));
    return error == hipSuccess;
}

void display()
{
    glUseProgram(prog);
    glBindTexture(GL_TEXTURE_2D, tex);
    glClear(GL_COLOR_BUFFER_BIT);
    glDrawArrays(GL_TRIANGLES, 0, 6);
}


__global__ void test(hipSurfaceObject_t surface, Scene *scene, hiprandState *state, float* data_tmp, int *cnt)
{
    __shared__ StratifiedSampler<TWO_FOR_SHARED> sampler;
    //if (threadIdx.x == 0 && blockIdx.x == 0 && blockIdx.y == 0)
    //    ++*cnt;
    __syncthreads();
    int idx = (blockIdx.x * blockDim.y + blockIdx.y) * 32 + threadIdx.x;


    //To do: use time to update the seed
    hiprand_init(3212, idx * 3,0, &state[idx]);

    if (threadIdx.x == 0)
    {
       sampler = StratifiedSampler<TWO_FOR_SHARED>(state + idx);
    }

    hiprandState *rstate = state + idx;

    Ray r;
    float3 tmp;
    int stx = blockIdx.x * WIDTH_PER_BLOCK, sty = blockIdx.y * HEIGHT_PER_BLOCK;

    StratifiedSampler<TWO> sampler_light(16, rstate);
    StratifiedSampler<TWO> sampler_surface(16, rstate);
    StratifiedSampler<ONE> p(8, rstate);

    float2 offset = make_float2(0.0f, 0.0f) ;
    int x, y;
    for(int i = 0; i < WIDTH_PER_BLOCK; i++)
        for (int j = 0; j < HEIGHT_PER_BLOCK; j++)
        {
            offset = sampler(threadIdx.x);
            r = globalCam.generateRay(i + stx + offset.x - WIDTH / 2, j + sty + offset.y - HEIGHT / 2);
            tmp = pathTracer(r, *scene, sampler_surface, sampler_light, p, rstate);

            idx = (j + sty) * WIDTH + i + stx;
            idx = idx * 3;
            data_tmp[idx] += tmp.x;
            data_tmp[idx + 1] += tmp.y;
            data_tmp[idx + 2] += tmp.z;
            
            sampler_surface.regenerate(rstate);
            sampler_light.regenerate(rstate);
            p.regenerate(rstate);
        }

    //__syncthreads();

    //idx = 3 * (sty * WIDTH + stx);
    //float weight = 1.0f / (32.0f * (*cnt));
    //Write to Texture
        //for (int i = threadIdx.x; i < WIDTH_PER_BLOCK; i+=32)
        //    for (int j = 0; j < HEIGHT_PER_BLOCK; j++)
        //    {
        //        x = i + stx;
        //        y = j + sty;
        //        idx = y * WIDTH + x;
        //        idx = idx * 3;
        //        
        //        //Bug here: should take average value
        //        surf2Dwrite(make_float4(data_tmp[idx] * weight,
        //            data_tmp[idx + 1] * weight, data_tmp[idx + 2] * weight, 1.0f), surface, sizeof(float4) * x , y);
        //    }
}

void test_for_initialize_scene()
{
    Scene scene;
    int lz[light::TYPE_NUM] = {1,0,0}, ms[model::TYPE_NUM] = {0,0,2};
    int mat_type[] = { material::LAMBERTIAN , material::LAMBERTIAN, material::LAMBERTIAN };
    Lambertian lamb(make_float3(0.7f, 0.8f, 0.4f)), lamb2(make_float3(0.8f, 0.0f, 0.0f)), lamb3(make_float3(1.0f, 1.0f, 1.0f));
    Material m(&lamb, material::LAMBERTIAN), c(&lamb2, material::LAMBERTIAN), cs(&lamb3, material::LAMBERTIAN);
    Material t[] = { m,c ,cs};
/*
    Triangle tria(
        make_float3(),
        );*/
    Quadratic q(make_float3(0.3f, 0.0f, 0.0f), Sphere);
    q.setUpTransformation(
        mat4(1.0f, 0.0f, 0.0f, 0.0f,
             0.0f, 1.0f, 0.0f, 0.0f,
            0.0f, 0.0f,1.0f, -10.0f,
            0.0f,0.0f,0.0f,1.0f)
    );

    Quadratic s(make_float3(1.0f, 0.0f, 0.0f), Sphere);
    s.setUpTransformation(
        mat4(1.0f, 0.0f, 0.0f, -2.0f,
            0.0f, 1.0f, 0.0f, -3.0f,
            0.0f, 0.0f, 1.0f, -5.0f,
            0.0f, 0.0f, 0.0f, 1.0f)
    );

    PointLight pl(make_float3(-8.0f, 0.0f, 0.0f), make_float3(300.0, 350.0f, 300.0f));
    Quadratic m_a[2] = { q,s};

    scene.initializeScene(
        lz, ms, &pl, nullptr, nullptr, nullptr, nullptr,
        m_a, mat_type, t
    );

    hipMalloc(&sce, sizeof(Scene));
    hipMemcpy(sce, &scene, sizeof(Scene), hipMemcpyHostToDevice);
//    cas<<<1,1>>>(sce);
}
#include "hip/hip_runtime.h"
#include "Scene.cuh"

#ifndef isBlack(x)
#define isBlack(x) (length((x)) < 0.001f)
#endif // !isBlack(x)

//Brute force method
CUDA_FUNC bool Scene::hit(Ray &r, IntersectRecord &rec) const
{
    for (int i = 0; i < model::TYPE_NUM; i++)
    {
        switch (model::MODEL_TYPE(i))
        {
        case model::TRIAGNLE:
            for (int j = 0; j < model_sz[i]; j++)
            {
                Triangle a = tri[j];
                a.hit(r, rec);
            }
            break;
        case model::MESH:
            for (int j = 0; j < model_sz[i]; j++)
            { 
                Mesh t = mesh[j]; 
                t.hit(r, rec);
            }
            break;
        case model::Quadratic:
            for (int j = 0; j < model_sz[i]; j++)
            {
                Quadratic q;
                q = quad[j];
                q.hit(r, rec);
            }
            break;
        default:
            break;
        }
    }

    for (int i = light::TRIANGLE_LIGHT; i < light::TYPE_NUM; i++)
    {
        switch (light::LIGHT_TYPE(i))
        {
        case light::TRIANGLE_LIGHT:
            for (int j = 0; j < light_sz[i]; j++)
            {
                TriangleLight l = tril[j];
                l.hit(r, rec);
            }
            break;
        case light::TYPE_NUM:
            break;
        default:
            break;
        }
    }

    return rec.t > 0 && rec.t < 100000.0f;
}

__device__ float3 Scene::sampleOneLight(IntersectRecord &rec, float2 sample_light, float2 sample_surface, int sample_num) const
{
    static unsigned int num, cnt;
    num = sample_num % light_sz_all;
    cnt = 0;

    Light *obj;
    PointLight pl;
    DirectionalLight dl;
    TriangleLight trl;

    for (unsigned int i = 0; i < (unsigned int)light::TYPE_NUM; i++)
    {
        if (num >= light_sz[i])
            num -= light_sz[i];
        else
        {
            switch (light::LIGHT_TYPE(i))
            {
            case light::POINT_LIGHT:
                pl =  pointl[num];
                obj = &pl;
                break;
            case light::DIR_LIGHT:
                dl =  dirl[num];
                obj = &dl;
                break;
            case light::TRIANGLE_LIGHT:
                trl =  tril[num];
                obj = &trl;
                break;
            default:
                obj =  nullptr;
                break;
            }
            break;
        }
    }

    return obj ? light_sz_all * inv_light_power * obj->getPower(bound2 - bound1) * evaluateDirectLight(obj, rec, sample_light, sample_surface) : make_float3(0.0f, 0.0f, 0.0f);

}

__host__ bool Scene::initializeScene(int light_size[], int model_size[], PointLight *point_light, 
    DirectionalLight *dir_light, TriangleLight *tri_light, Triangle *triangles, Mesh *meshes, Quadratic *quadratic)
{
    hipError_t error;
    
    error = hipMalloc(&pointl, sizeof(PointLight) * light_size[0]);
    error = hipMalloc(&dirl, sizeof(DirectionalLight) * light_size[1]);
    error = hipMalloc(&pointl, sizeof(TriangleLight) * light_size[2]);
    //error = hipMalloc(&pointl, sizeof(PointLight) * light_size[3]);
    
    error = hipMalloc(&tri, sizeof(Triangle) * model_size[0]);
    error = hipMalloc(&mesh, sizeof(Mesh) * model_size[1]);
    error = hipMalloc(&quad, sizeof(Quadratic) * model_size[2]);
      
    error = hipMemcpy(pointl, point_light, sizeof(PointLight) * light_size[0], hipMemcpyHostToDevice);
    error = hipMemcpy(dirl, dir_light, sizeof(DirectionalLight) * light_size[1], hipMemcpyHostToDevice);
    error = hipMemcpy(tril, tri_light, sizeof(TriangleLight) * light_size[2], hipMemcpyHostToDevice);

    error = hipMemcpy(tri, triangles, sizeof(Triangle) * model_size[0], hipMemcpyHostToDevice);
    error = hipMemcpy(mesh, meshes, sizeof(Mesh) * model_size[1], hipMemcpyHostToDevice);
    error = hipMemcpy(quad, quadratic, sizeof(Quadratic) * model_size[2], hipMemcpyHostToDevice);

    model_sz_all = 0;
    for (int i = 0; i < model::TYPE_NUM; i++)
        model_sz_all += model_size[i];
    
    light_sz_all = 0;
    inv_light_power = 0.0f;
    for (unsigned int i = 0; i < (unsigned int)light::TYPE_NUM; i++)
    {
        light_sz_all += light_sz[i];
        
        int j;
        switch (light::LIGHT_TYPE(i))
        {
        case light::POINT_LIGHT:
            for (j = 0; j < light_sz[i]; j++)
                inv_light_power += point_light[j].getPower();
            break;
        case light::DIR_LIGHT:
            for (j = 0; j < light_sz[i]; j++)
                inv_light_power += dir_light[j].getPower(bound2 - bound1);
            break;
        case light::TRIANGLE_LIGHT:
            for (j = 0; j < light_sz[i]; j++)
                inv_light_power += tri_light[j].getPower();
            break;
        default:
            break;
        }
    }

    inv_light_power = 1.0f / inv_light_power;

    return error == hipSuccess;
}

//Incompleted
CUDA_FUNC float3 Scene::evaluateDirectLight(Light *light, IntersectRecord &rec, float2 sample_light, float2 sample_BRDF) const
{
    Ray r;
    float distance;
    float3 color, res;
    bool blocked = false;

    color = light->lightIllumi(rec, &r, sample_light);
    distance = length(rec.pos - r.getOrigin());


    Material *this_material = rec.material;

    IntersectRecord light_rec;
    hit(r, light_rec);

    Ray wo;
    float3 f = this_material -> f(rec.wo.getDir(), -r.getDir());

    if (light_rec.t < distance - 0.001f)
        blocked = true;

    res = BLACK;
    if (!isBlack(f) && !blocked)
    {
        if (light->isDelta)
        {
            color = fabs(dot(rec.normal, r.getDir())) * color * f / rec.pdf_light;
        }
        else
        {
            rec.pdf_surface = this_material->PDF(wo.getDir(), r.getDir());
            color = fabs(dot(rec.normal, r.getDir())) * color * f
                *  PowerHeuristic(rec.pdf_surface, rec.pdf_light) / rec.pdf_light;
        }
        res = color;
    }
    float3 wi;
    
    if (!light->isDelta)
    {
        f = this_material->sample_f(wo.getDir(), &wi, &rec.pdf_surface, sample_BRDF);
        f *= fabs(dot(rec.normal, wi));
        r = Ray(r.getOrigin(), wi);
        //r = rec.spawnRay(r);

        rec.t = 100000.0f;
        rec.light = nullptr;
        if (!isBlack(f) && rec.pdf_surface > 0.00001f)
        {
            float pdf = light->PDF(rec, wi);
            if (fabs(pdf) > 0.00001f)
            {
                float3 l;
                if (hit(r, rec))
                {
                    if (rec.light == (void*)light)
                        l = light ->getLe(Ray(r.getOrigin(), -wi) , &rec);
                    else
                    {
                        Light *a =(Light*) rec.light;
                        l = a->getLe(r, &rec);
                    }

                    res += l * f * PowerHeuristic(rec.pdf_surface, rec.pdf_light) / rec.pdf_surface;
                }
            }

            return res;
        }
    }

    return res;
}
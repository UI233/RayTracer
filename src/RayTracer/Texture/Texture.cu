#include "hip/hip_runtime.h"
#include "Texture.cuh"
uint getMipMapLevels(hipExtent size) {
	size_t sz = MAX(MAX(size.width, size.height), size.depth);

	uint levels = 0;

	while (sz) {
		sz /= 2;
		levels++;
	}

	return levels;
}
__global__ void
d_mipmap(hipSurfaceObject_t mipOutput, hipTextureObject_t mipInput, float4* test, uint imageW, uint imageH, uint threadWidth, uint threadHeight) {
	uint x = (blockIdx.x * blockDim.x + threadIdx.x)*threadWidth;
	uint y = (blockIdx.y * blockDim.y + threadIdx.y)*threadHeight;

	float px = 1.0 / float(imageW);
	float py = 1.0 / float(imageH);

	for (int i = 0; i < threadHeight; i++) {
		for (int j = 0; j < threadWidth; j++) {
			if ((x + j < imageW) && (y + i < imageH)) {
				// take the average of 4 samples

				// we are using the normalized access to make sure non-power-of-two textures
				// behave well when downsized.
				float4 color =
					(tex2D<float4>(mipInput, (x + j + 0) * px, (y + i + 0) * py)) +
					(tex2D<float4>(mipInput, (x + j + 1) * px, (y + i + 0) * py)) +
					(tex2D<float4>(mipInput, (x + j + 1) * px, (y + i + 1) * py)) +
					(tex2D<float4>(mipInput, (x + j + 0) * px, (y + i + 1) * py));


				color /= 4.0;
				color = fminf(color, make_float4(1.0));
				//printf("%f %f %f %f\n", color.x, color.y, color.z, color.w);
				surf2Dwrite(color, mipOutput, (x + j) * sizeof(float4), y + i);
				int idx = (y + i) * imageW + j + x;
				test[idx] = color;
			}
		}
	}
}


void generateMipMaps(hipMipmappedArray_t mipmapArray, hipExtent size) {
	size_t width = size.width;
	size_t height = size.height;

	uint level = 0;

	while (width != 1 || height != 1) {
		width /= 2;
		width = MAX((size_t)1, width);
		height /= 2;
		height = MAX((size_t)1, height);
		std::cout << width << " " << height << std::endl;
		hipArray_t levelFrom;
		auto error = hipGetMipmappedArrayLevel(&levelFrom, mipmapArray, level);
		hipArray_t levelTo;
		error = hipGetMipmappedArrayLevel(&levelTo, mipmapArray, level + 1);

		hipExtent  levelToSize;
		error = hipArrayGetInfo(NULL, &levelToSize, NULL, levelTo);
		levelToSize.width == width;
		levelToSize.height == height;
		levelToSize.depth == 0;

		// generate texture object for reading
		hipTextureObject_t         texInput;
		hipResourceDesc            texRes;
		memset(&texRes, 0, sizeof(hipResourceDesc));

		texRes.resType = hipResourceTypeArray;
		texRes.res.array.array = levelFrom;

		hipTextureDesc             texDescr;
		memset(&texDescr, 0, sizeof(hipTextureDesc));

		texDescr.normalizedCoords = 1;
		texDescr.filterMode = hipFilterModeLinear;

		texDescr.addressMode[0] = hipAddressModeClamp;
		texDescr.addressMode[1] = hipAddressModeClamp;
		texDescr.addressMode[2] = hipAddressModeClamp;

		texDescr.readMode = hipReadModeElementType;;
		float4 * test;
		error = hipMalloc(&test, width*height * sizeof(float4));
		error = hipCreateTextureObject(&texInput, &texRes, &texDescr, NULL);

		// generate surface object for writing

		hipSurfaceObject_t surfOutput;
		hipResourceDesc    surfRes;
		memset(&surfRes, 0, sizeof(hipResourceDesc));
		surfRes.resType = hipResourceTypeArray;
		surfRes.res.array.array = levelTo;

		error = hipCreateSurfaceObject(&surfOutput, &surfRes);

		// run mipmap kernel
		dim3 blockSize(16, 16, 1);
		dim3 gridSize(((uint)width + blockSize.x - 1) / blockSize.x, ((uint)height + blockSize.y - 1) / blockSize.y, 1);

		d_mipmap << <gridSize, blockSize >> > (surfOutput, texInput, test, (uint)width, (uint)height, 1, 1);

		error = hipDeviceSynchronize();
		float4 *output = (float4 *)malloc(width*height * sizeof(float4));
		error = hipMemcpy(output, test, width*height * sizeof(float4), hipMemcpyDeviceToHost);
			std::string a=".ppm";
			a = std::to_string(width)+a;

			std::ofstream fp(a, std::ios::out);
			fp << "P3" << std::endl;
			fp << width << " " << height << std::endl;
			fp << "255" << std::endl;
			for (int i = 0; i < height; i++) {
				for (int j = 0; j < width; j++) {
					fp << int(output[i*width + j].x*255) << " " << int(output[i*width + j].y*255) << " " << int(output[i*width + j].z*255) << std::endl;
				}
			}
		error = hipGetLastError();
		fp.close();
		hipDestroySurfaceObject(surfOutput);

		hipDestroyTextureObject(texInput);
		free(output);
		level++;
	}
}

void initImages(Image *images, float *data, int width, int height) {
	// create individual textures
	Image &image = *images;
	float highestLod = 0;
	image.h_data = data;
	std::cout << width << height << std::endl;
	image.size = make_hipExtent(width, height, 0);
	image.size.depth = 0;
	image.type = hipResourceTypeMipmappedArray;
	// how many mipmaps we need
	uint levels = getMipMapLevels(image.size);
	highestLod = MAX(highestLod, (float)levels - 1);
	hipChannelFormatDesc desc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
	hipMallocMipmappedArray(&image.mipmapArray, &desc, image.size, levels);
	// upload level 0
	hipArray_t level0;
	hipGetMipmappedArrayLevel(&level0, image.mipmapArray, 0);

	hipMemcpy3DParms copyParams = { 0 };
	copyParams.srcPtr = make_hipPitchedPtr(image.h_data, image.size.width * sizeof(float4), image.size.width, image.size.height);
	copyParams.dstArray = level0;
	copyParams.extent = image.size;
	copyParams.extent.depth = 1;
	copyParams.kind = hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);
	// compute rest of mipmaps based on level 0
	generateMipMaps(image.mipmapArray, image.size);

	// generate bindless texture object

	hipResourceDesc            resDescr;
	memset(&resDescr, 0, sizeof(hipResourceDesc));

	resDescr.resType = hipResourceTypeMipmappedArray;
	resDescr.res.mipmap.mipmap = image.mipmapArray;

	hipTextureDesc             texDescr;
	memset(&texDescr, 0, sizeof(hipTextureDesc));

	texDescr.normalizedCoords = 1;
	texDescr.filterMode = hipFilterModeLinear;
	texDescr.mipmapFilterMode = hipFilterModeLinear;
	texDescr.addressMode[0] = hipAddressModeClamp;
	texDescr.addressMode[1] = hipAddressModeClamp;
	texDescr.addressMode[2] = hipAddressModeClamp;

	texDescr.maxMipmapLevelClamp = float(levels - 1);
	texDescr.readMode = hipReadModeElementType;

	auto error = hipCreateTextureObject(&image.textureObject, &resDescr, &texDescr, NULL);

}
